#include "hip/hip_runtime.h"
#include <vector>

#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_segmented_radix_sort.cuh>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <faiss/gpu/utils/BlockSelectKernel.cuh>
#include <faiss/gpu/utils/DeviceTensor.cuh>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>

#include "implicit/gpu/knn.h"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {
namespace {
const static int TILE_GROUPS = 32;
const static int MAX_TILE_ROWS = 32;

// faiss seems to have issues when distances contain -FLT_MAX, and can return a
// '-1' in the indices returned, instead of an actual valid row number. When we
// filter, instead of setting to -FLT_MAX, set to the next smallest valid
// float32 value.
const static float _FLT_MAX = FLT_MAX;
const static uint32_t UINT_FILTER_DISTANCE =
    (*reinterpret_cast<const uint32_t *>(&_FLT_MAX)) - 1;
const static float FLT_FILTER_DISTANCE =
    -*reinterpret_cast<const float *>(&UINT_FILTER_DISTANCE);
} // namespace

bool is_host_memory(void *address) {
  hipPointerAttribute_t attr;
  auto err = hipPointerGetAttributes(&attr, address);
  if (err == hipErrorInvalidValue) {
    return true;
  }

#if __CUDACC_VER_MAJOR__ >= 10
  return attr.type == hipMemoryTypeHost ||
         attr.type == cudaMemoryTypeUnregistered;
#else
  return attr.memoryType == hipMemoryTypeHost ||
         attr.memoryType == cudaMemoryTypeUnregistered;
#endif
}

template <typename T>
void copy_columns(const T *input, int rows, int cols, T *output,
                  int output_cols) {
  auto count = thrust::make_counting_iterator<int>(0);
  thrust::for_each(count, count + (rows * output_cols), [=] __device__(int i) {
    int col = i % output_cols;
    int row = i / output_cols;
    output[col + row * output_cols] = input[col + row * cols];
  });
}

KnnQuery::KnnQuery(size_t temp_memory) {
  if (!temp_memory) {
    // use half of free GPU memory, limited to 8GB max
    size_t free, total;
    CHECK_CUDA(hipMemGetInfo(&free, &total));
    temp_memory = std::min(free / 2, static_cast<size_t>(8000000000));
  }

  // pad out to 256 bytes if necessary
  size_t padding = temp_memory % 256;
  if (padding) {
    temp_memory += 256 - padding;
  }

  max_temp_memory = temp_memory;

  static rmm::mr::cuda_memory_resource upstream_mr;
  mr.reset(new rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource>(
      &upstream_mr, max_temp_memory, max_temp_memory));

  CHECK_CUBLAS(hipblasCreate(&blas_handle));
}

void KnnQuery::topk(const Matrix &items, const Matrix &query, int k,
                    int *indices, float *distances, float *item_norms,
                    const COOMatrix *query_filter, Vector<int> *item_filter) {
  if (query.cols != items.cols) {
    throw std::invalid_argument(
        "Must have same number of columns in each matrix for topk");
  }

  rmm::cuda_stream_view stream;

  size_t available_temp_memory = max_temp_memory;

  // limit to temp memory 8GB or so (causes some issues if we have over 2^31
  // entries in our matrix
  available_temp_memory =
      std::min(available_temp_memory, static_cast<size_t>(8000000000));

  float *host_distances = NULL;
  std::unique_ptr<rmm::device_uvector<float>> distances_storage;
  size_t distances_size = query.rows * k * sizeof(float);
  if (is_host_memory(distances)) {
    host_distances = distances;
    distances_storage.reset(
        new rmm::device_uvector<float>(query.rows * k, stream, mr.get()));
    distances = distances_storage->data();
    available_temp_memory -= distances_size;
  }

  int *host_indices = NULL;
  std::unique_ptr<rmm::device_uvector<int>> indices_storage;
  size_t indices_size = query.rows * k * sizeof(int);
  if (is_host_memory(indices)) {
    host_indices = indices;
    indices_storage.reset(
        new rmm::device_uvector<int>(query.rows * k, stream, mr.get()));
    indices = indices_storage->data();
    available_temp_memory -= indices_size;
  }

  // Create temporary memory for storing results. We're padding out temp memory
  // so that we can tile the columns (break up a single row to multiple top-k
  // operations) if there aren't many rows in the input
  size_t temp_distances_cols = items.rows;
  size_t padding = 0;

  // just in case we're tiling each row, we'l need some temp memory for that too
  size_t tile_memory =
      TILE_GROUPS * MAX_TILE_ROWS * k * (sizeof(float) + sizeof(int));
  bool allow_tiling = tile_memory * 4 < available_temp_memory;
  if (allow_tiling) {
    padding = temp_distances_cols % TILE_GROUPS;
    if (padding) {
      temp_distances_cols += TILE_GROUPS - padding;
    }
    available_temp_memory -= tile_memory;
  }

  // We need 6 copies of the matrix for argsort code - and then some
  // extra memory per SM as well.
  size_t batch_size =
      (available_temp_memory /
       (sizeof(float) * static_cast<size_t>(temp_distances_cols)));
  if (k >= GPU_MAX_SELECTION_K) {
    batch_size *= 0.15;
  }

  batch_size = std::min(batch_size, query.rows);
  batch_size = std::max(batch_size, static_cast<size_t>(1));

  rmm::device_uvector<float> temp_mem(batch_size * temp_distances_cols, stream,
                                      mr.get());
  Matrix temp_distances(batch_size, temp_distances_cols, temp_mem.data(),
                        false);

  // Fill temp_distances if we're padding so that results don't appear
  if (padding) {
    thrust::device_ptr<float> data =
        thrust::device_pointer_cast(temp_distances.data);
    thrust::fill(data, data + temp_distances.rows * temp_distances.cols,
                 -FLT_MAX);
  }

  for (int start = 0; start < query.rows; start += batch_size) {
    auto end = std::min(query.rows, start + batch_size);

    Matrix batch(query, start, end);
    temp_distances.rows = batch.rows;

    // matrix multiple the items by the batch, store in distances
    float alpha = 1.0, beta = 0.;

    CHECK_CUBLAS(hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items.rows,
                             batch.rows, items.cols, &alpha, items.data,
                             items.cols, batch.data, batch.cols, &beta,
                             temp_distances.data, temp_distances.cols));

    // If we have norms (cosine distance etc) normalize the results here
    if (item_norms != NULL) {
      auto count = thrust::make_counting_iterator<size_t>(0);
      int cols = temp_distances.cols;
      int item_norm_cols = items.rows;
      float *data = temp_distances.data;
      thrust::for_each(count,
                       count + (static_cast<size_t>(temp_distances.rows) *
                                static_cast<size_t>(temp_distances.cols)),
                       [=] __device__(size_t i) {
                         int col = i % cols;
                         if (col < item_norm_cols) {
                           data[i] /= item_norms[col];
                         }
                       });
    }

    if (item_filter != NULL) {
      auto count = thrust::make_counting_iterator<size_t>(0);
      float *data = temp_distances.data;
      int *items = item_filter->data;
      int items_size = item_filter->size;
      int cols = temp_distances.cols;
      float filter_distance = FLT_FILTER_DISTANCE;
      thrust::for_each(count, count + items_size * temp_distances.rows,
                       [=] __device__(size_t i) {
                         int col = items[i % items_size];
                         int row = i / items_size;
                         data[row * cols + col] = filter_distance;
                       });
    }

    if (query_filter != NULL) {
      auto count = thrust::make_counting_iterator<size_t>(0);
      int *row = query_filter->row;
      int *col = query_filter->col;
      float *data = temp_distances.data;
      int items = temp_distances.cols;
      float filter_distance = FLT_FILTER_DISTANCE;
      thrust::for_each(
          count, count + query_filter->nonzeros, [=] __device__(int i) {
            if ((row[i] >= start) && (row[i] < end)) {
              data[(row[i] - start) * items + col[i]] = filter_distance;
            }
          });
    }

    argpartition(temp_distances, k, indices + start * k, distances + start * k,
                 allow_tiling);

    // TODO: callback per batch (show progress etc)
  }

  if (host_indices) {
    CHECK_CUDA(hipMemcpy(host_indices, indices, indices_size,
                          hipMemcpyDeviceToHost));
  }

  if (host_distances) {
    CHECK_CUDA(hipMemcpy(host_distances, distances, distances_size,
                          hipMemcpyDeviceToHost));
  }
}

void KnnQuery::argpartition(const Matrix &items, int k, int *indices,
                            float *distances, bool allow_tiling) {
  k = std::min(k, static_cast<int>(items.cols));

  if (k >= GPU_MAX_SELECTION_K) {
    rmm::cuda_stream_view stream;
    rmm::device_uvector<int> temp_indices(items.rows * items.cols, stream,
                                          mr.get());
    rmm::device_uvector<float> temp_distances(items.rows * items.cols, stream,
                                              mr.get());
    argsort(items, temp_indices.data(), temp_distances.data());
    copy_columns(temp_distances.data(), items.rows, items.cols, distances, k);
    copy_columns(temp_indices.data(), items.rows, items.cols, indices, k);
    return;
  }

  int rows = items.rows;
  int cols = items.cols;

  // faiss runBlockSelect isn't the fastest option when there aren't that many
  // rows, since each row in the query only gets a single thread block to
  // process it. For queries with a small number of rows, we're going to break
  // up each row into TILE_GROUPS sub-rows, in one runBlockSelect, and then
  // combine the results from those in a final select op.
  bool tile_rows =
      (rows <= MAX_TILE_ROWS) && (cols % TILE_GROUPS == 0) && (cols >= 65536);
  if (allow_tiling && tile_rows) {
    // Run the first block select on the sub-rows
    int rows_tile = rows * TILE_GROUPS;
    int cols_tile = cols / TILE_GROUPS;

    rmm::cuda_stream_view stream;
    rmm::device_uvector<int> temp_indices(rows_tile * k, stream, mr.get());
    rmm::device_uvector<float> temp_distances(rows_tile * k, stream, mr.get());

    faiss::gpu::DeviceTensor<float, 2, true> items_tensor(
        const_cast<float *>(items.data), {rows_tile, cols_tile});
    faiss::gpu::DeviceTensor<float, 2, true> temp_distances_tensor(
        temp_distances.data(), {rows_tile, k});
    faiss::gpu::DeviceTensor<int, 2, true> temp_indices_tensor(
        temp_indices.data(), {rows_tile, k});
    faiss::gpu::runBlockSelect(items_tensor, temp_distances_tensor,
                               temp_indices_tensor, true, k, 0);

    // Calculate the true index for all the topk results (since the current
    // temp_indices will be relative to the split values)
    auto count = thrust::make_counting_iterator<size_t>(0);
    int *temp_indices_ptr = temp_indices.data();
    thrust::for_each(count, count + rows_tile * k, [=] __device__(int i) {
      int offset = cols_tile * ((i / k) % TILE_GROUPS);
      temp_indices_ptr[i] += offset;
    });

    // reshape the temp tensors we calculated in the first pass, and then get
    // the actual output
    faiss::gpu::DeviceTensor<float, 2, true> temp_input_distances_tensor(
        temp_distances.data(), {rows, k * TILE_GROUPS});
    faiss::gpu::DeviceTensor<int, 2, true> temp_input_indices_tensor(
        temp_indices.data(), {rows, k * TILE_GROUPS});
    faiss::gpu::DeviceTensor<float, 2, true> distances_tensor(distances,
                                                              {rows, k});
    faiss::gpu::DeviceTensor<int, 2, true> indices_tensor(indices, {rows, k});
    faiss::gpu::runBlockSelectPair(temp_input_distances_tensor,
                                   temp_input_indices_tensor, distances_tensor,
                                   indices_tensor, true, k, 0);
  } else {
    faiss::gpu::DeviceTensor<float, 2, true> items_tensor(
        const_cast<float *>(items.data), {rows, cols});
    faiss::gpu::DeviceTensor<float, 2, true> distances_tensor(distances,
                                                              {rows, k});
    faiss::gpu::DeviceTensor<int, 2, true> indices_tensor(indices, {rows, k});
    faiss::gpu::runBlockSelect(items_tensor, distances_tensor, indices_tensor,
                               true, k, 0);
  }

  CHECK_CUDA(hipDeviceSynchronize());
}

void KnnQuery::argsort(const Matrix &items, int *indices, float *distances) {
  // We can't do this in place https://github.com/NVIDIA/cub/issues/238 ?
  // so generate temp memory for this

  rmm::cuda_stream_view stream;

  rmm::device_uvector<int> temp_indices(items.rows * items.cols, stream,
                                        mr.get());
  thrust::transform(
      thrust::make_counting_iterator<int>(0),
      thrust::make_counting_iterator<int>(items.rows * items.cols),
      thrust::make_constant_iterator<int>(items.cols),
      thrust::device_pointer_cast(temp_indices.data()), thrust::modulus<int>());

  int cols = items.cols;
  auto segment_offsets = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int>(0),
      [=] __device__(int i) { return i * cols; });

  void *temp_mem = NULL;
  size_t temp_size = 0;

  // sort the values.
  if (items.rows > 1) {
    auto err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        NULL, temp_size, items.data, distances, temp_indices.data(), indices,
        items.rows * items.cols, items.rows, segment_offsets,
        segment_offsets + 1);
    CHECK_CUDA(err);
    temp_mem = mr->allocate(temp_size, stream);
    err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        temp_mem, temp_size, items.data, distances, temp_indices.data(),
        indices, items.rows * items.cols, items.rows, segment_offsets,
        segment_offsets + 1);
    CHECK_CUDA(err);
  } else {
    size_t temp_size = 0;
    auto err = hipcub::DeviceRadixSort::SortPairsDescending(
        NULL, temp_size, items.data, distances, temp_indices.data(), indices,
        items.cols);
    CHECK_CUDA(err);
    temp_mem = mr->allocate(temp_size, stream);
    err = hipcub::DeviceRadixSort::SortPairsDescending(
        temp_mem, temp_size, items.data, distances, temp_indices.data(),
        indices, items.cols);
    CHECK_CUDA(err);
  }
  mr->deallocate(temp_mem, temp_size, stream);
}

KnnQuery::~KnnQuery() {
  // TODO: don't check this, there isn't anything we can do here anyways
  CHECK_CUBLAS(hipblasDestroy(blas_handle));
}

} // namespace gpu
} // namespace implicit
