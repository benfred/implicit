#include "hip/hip_runtime.h"
#include <vector>

#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_segmented_radix_sort.cuh>
#include <hip/hip_runtime.h>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>

#include <raft/core/resource/cublas_handle.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resource/thrust_policy.hpp>
#include <raft/matrix/select_k.cuh>

#include "implicit/gpu/knn.h"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {
namespace {
// faiss seems to have issues when distances contain -FLT_MAX, and can return a
// '-1' in the indices returned, instead of an actual valid row number. When we
// filter, instead of setting to -FLT_MAX, set to the next smallest valid
// float32 value.
const static float _FLT_MAX = FLT_MAX;
const static uint32_t UINT_FILTER_DISTANCE =
    (*reinterpret_cast<const uint32_t *>(&_FLT_MAX)) - 1;
const static float FLT_FILTER_DISTANCE =
    -*reinterpret_cast<const float *>(&UINT_FILTER_DISTANCE);
} // namespace

bool is_host_memory(void *address) {
  hipPointerAttribute_t attr;
  auto err = hipPointerGetAttributes(&attr, address);
  if (err == hipErrorInvalidValue) {
    return true;
  }

#if __CUDACC_VER_MAJOR__ >= 10
  return attr.type == hipMemoryTypeHost ||
         attr.type == cudaMemoryTypeUnregistered;
#else
  return attr.memoryType == hipMemoryTypeHost ||
         attr.memoryType == cudaMemoryTypeUnregistered;
#endif
}

KnnQuery::KnnQuery(size_t temp_memory) {
  if (!temp_memory) {
    // use half of free GPU memory, limited to 4GB max
    size_t free, total;
    CHECK_CUDA(hipMemGetInfo(&free, &total));
    temp_memory = std::min(free / 2, size_t{4294967296});
  }

  // pad out to 256 bytes if necessary
  size_t padding = temp_memory % 256;
  if (padding) {
    temp_memory += 256 - padding;
  }

  max_temp_memory = temp_memory;

  static rmm::mr::cuda_memory_resource upstream_mr;
  mr.reset(new rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource>(
      &upstream_mr, max_temp_memory, max_temp_memory));
}

void KnnQuery::topk(const Matrix &items, const Matrix &query, int k,
                    int *indices, float *distances, Matrix *item_norms,
                    const COOMatrix *query_filter, Vector<int> *item_filter) {
  if (query.cols != items.cols) {
    throw std::invalid_argument(
        "Must have same number of columns in each matrix for topk");
  }

  if (query.itemsize != items.itemsize) {
    throw std::invalid_argument("Must have same dtype in each matrix for topk");
  }

  if (items.itemsize == 4) {
    topk_impl<float>(items, query, k, indices, distances, item_norms,
                     query_filter, item_filter);
  } else if (items.itemsize == 2) {
    topk_impl<half>(items, query, k, indices, distances, item_norms,
                    query_filter, item_filter);
  } else {
    throw std::invalid_argument("invalid itemsize for topk");
  }
}

namespace {

template <typename T>
void gemm(cublasContext *blas_handle, const Matrix &items, const Matrix &batch,
          Matrix *output);

template <>
void gemm<float>(cublasContext *blas_handle, const Matrix &items,
                 const Matrix &batch, Matrix *output) {
  // matrix multiple the items by the batch, store in distances
  float alpha = 1.0, beta = 0.;

  CHECK_CUBLAS(hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items.rows,
                           batch.rows, items.cols, &alpha, items, items.cols,
                           batch, batch.cols, &beta, *output, output->cols));
}

template <>
void gemm<half>(cublasContext *blas_handle, const Matrix &items,
                const Matrix &batch, Matrix *output) {
  // matrix multiple the items by the batch, store in distances
  float alpha = 1.0, beta = 0.;

  // our factors are float16, but we accumulate into a float32
  CHECK_CUBLAS(cublasSgemmEx(
      blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, items.rows, batch.rows, items.cols,
      &alpha, items.data, HIP_R_16F, items.cols, batch.data, HIP_R_16F,
      batch.cols, &beta, output->data, HIP_R_32F, output->cols));
}
} // namespace

template <typename T>
void KnnQuery::topk_impl(const Matrix &items, const Matrix &query, int k,
                         int *indices, float *distances, Matrix *item_norms,
                         const COOMatrix *query_filter,
                         Vector<int> *item_filter) {
  auto stream = raft::resource::get_cuda_stream(handle);

  size_t available_temp_memory = max_temp_memory;

  // limit to temp memory 8GB or so (causes some issues if we have over 2^31
  // entries in our matrix)
  available_temp_memory = std::min(available_temp_memory, size_t{4294967296});

  float *host_distances = NULL;
  std::unique_ptr<rmm::device_uvector<float>> distances_storage;
  size_t distances_size = query.rows * k * sizeof(float);
  if (is_host_memory(distances)) {
    host_distances = distances;
    distances_storage.reset(
        new rmm::device_uvector<float>(query.rows * k, stream, mr.get()));
    distances = distances_storage->data();
    available_temp_memory -= distances_size;
  }

  int *host_indices = NULL;
  std::unique_ptr<rmm::device_uvector<int>> indices_storage;
  size_t indices_size = query.rows * k * sizeof(int);
  if (is_host_memory(indices)) {
    host_indices = indices;
    indices_storage.reset(
        new rmm::device_uvector<int>(query.rows * k, stream, mr.get()));
    indices = indices_storage->data();
    available_temp_memory -= indices_size;
  }

  // extra temp memory for sorting the output
  available_temp_memory -= (query.rows * k * (sizeof(float) + sizeof(int))) * 3;

  size_t batch_size = (available_temp_memory /
                       (sizeof(float) * static_cast<size_t>(items.rows)));

  batch_size = std::min(batch_size, query.rows);
  batch_size = std::max(batch_size, static_cast<size_t>(1));

  rmm::device_uvector<T> temp_mem(batch_size * items.rows, stream, mr.get());
  Matrix temp_distances(batch_size, items.rows, temp_mem.data(), false);

  for (int start = 0; start < query.rows; start += batch_size) {
    auto end = std::min(query.rows, start + batch_size);

    Matrix batch(query, start, end);
    temp_distances.rows = batch.rows;

    auto blas_handle = raft::resource::get_cublas_handle(handle);
    gemm<T>(blas_handle, items, batch, &temp_distances);

    auto thrust_policy = raft::resource::get_thrust_policy(handle);
    // If we have norms (cosine distance etc) normalize the results here
    if (item_norms != NULL) {
      auto count = thrust::make_counting_iterator<size_t>(0);
      int cols = temp_distances.cols;
      int item_norm_cols = items.rows;
      float *data = temp_distances;
      const float *item_norms_data = *item_norms;
      thrust::for_each(thrust_policy, count,
                       count + (static_cast<size_t>(temp_distances.rows) *
                                static_cast<size_t>(temp_distances.cols)),
                       [=] __device__(size_t i) {
                         int col = i % cols;
                         if (col < item_norm_cols) {
                           data[i] /= item_norms_data[col];
                         }
                       });
    }

    if (item_filter != NULL) {
      auto count = thrust::make_counting_iterator<size_t>(0);
      float *data = temp_distances;
      int *items = item_filter->data;
      int items_size = item_filter->size;
      int cols = temp_distances.cols;

      float filter_distance = FLT_FILTER_DISTANCE;
      thrust::for_each(thrust_policy, count,
                       count + items_size * temp_distances.rows,
                       [=] __device__(size_t i) {
                         int col = items[i % items_size];
                         int row = i / items_size;
                         data[row * cols + col] = filter_distance;
                       });
    }

    if (query_filter != NULL) {
      auto count = thrust::make_counting_iterator<size_t>(0);
      int *row = query_filter->row;
      int *col = query_filter->col;
      float *data = temp_distances;
      int items = temp_distances.cols;
      float filter_distance = FLT_FILTER_DISTANCE;
      thrust::for_each(thrust_policy, count, count + query_filter->nonzeros,
                       [=] __device__(int i) {
                         if ((row[i] >= start) && (row[i] < end)) {
                           data[(row[i] - start) * items + col[i]] =
                               filter_distance;
                         }
                       });
    }

    auto current_k = std::min(k, static_cast<int>(temp_distances.cols));
    rmm::device_uvector<float> best_distances(temp_distances.rows * current_k,
                                              stream, mr.get());
    rmm::device_uvector<int> best_indices(temp_distances.rows * current_k,
                                          stream, mr.get());

    auto distance_view = raft::make_device_matrix_view<const float, int64_t>(
        temp_distances, temp_distances.rows, temp_distances.cols);

    raft::matrix::select_k<float, int>(
        handle, distance_view, std::nullopt,
        raft::make_device_matrix_view<float, int64_t>(
            best_distances.data(), temp_distances.rows, current_k),
        raft::make_device_matrix_view<int, int64_t>(
            best_indices.data(), temp_distances.rows, current_k),
        false);

    // raft::select_k doesn't sort inputs - so we have to do it here
    argsort(best_indices.data(), best_distances.data(), temp_distances.rows,
            current_k, indices + start * k, distances + start * k);
    // TODO: callback per batch (show progress etc)
  }

  raft::resource::sync_stream(handle);

  if (host_indices) {
    CHECK_CUDA(hipMemcpy(host_indices, indices, indices_size,
                          hipMemcpyDeviceToHost));
  }

  if (host_distances) {
    CHECK_CUDA(hipMemcpy(host_distances, distances, distances_size,
                          hipMemcpyDeviceToHost));
  }
}

void KnnQuery::argsort(const int *input_indices, const float *input_distances,
                       int rows, int cols, int *indices, float *distances) {
  rmm::cuda_stream_view stream;
  auto segment_offsets = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int>(0),
      [=] __device__(int i) { return i * cols; });

  void *temp_mem = NULL;
  size_t temp_size = 0;

  // sort the values.
  if (rows > 1) {
    auto err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        NULL, temp_size, input_distances, distances, input_indices, indices,
        rows * cols, rows, segment_offsets, segment_offsets + 1, 0,
        sizeof(float) * 8, stream);
    CHECK_CUDA(err);
    temp_mem = mr->allocate(temp_size, stream);
    err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        temp_mem, temp_size, input_distances, distances, input_indices, indices,
        rows * cols, rows, segment_offsets, segment_offsets + 1, 0,
        sizeof(float) * 8, stream);
    CHECK_CUDA(err);
  } else {
    size_t temp_size = 0;
    auto err = hipcub::DeviceRadixSort::SortPairsDescending(
        NULL, temp_size, input_distances, distances, input_indices, indices,
        cols, 0, sizeof(float) * 8, stream);
    CHECK_CUDA(err);
    temp_mem = mr->allocate(temp_size, stream);
    err = hipcub::DeviceRadixSort::SortPairsDescending(
        temp_mem, temp_size, input_distances, distances, input_indices, indices,
        cols, 0, sizeof(float) * 8, stream);
    CHECK_CUDA(err);
  }
  mr->deallocate(temp_mem, temp_size, stream);
}

KnnQuery::~KnnQuery() {}

} // namespace gpu
} // namespace implicit
