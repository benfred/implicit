#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>


#include "implicit/gpu/matrix.h"
#include "implicit/gpu/utils.cuh"

namespace implicit { namespace gpu {
// TODO: this class doesn't seem to offer much on top of devicebuffer itself
template <typename T>
Vector<T>::Vector(int size, const T * host_data)
    : size(size), storage(new DeviceBuffer<T>(size)), data(storage->get()) {
    if (host_data) {
        CHECK_CUDA(hipMemcpy(data, host_data, size * sizeof(T), hipMemcpyHostToDevice));
    }
}

template <typename T>
void Vector<T>::to_host(T * out) const {
    CHECK_CUDA(hipMemcpy(out, data, size * sizeof(T), hipMemcpyDeviceToHost));
}

template struct Vector<char>;
template struct Vector<int>;
template struct Vector<float>;

Matrix::Matrix(const Matrix & other, int rowid)
    : rows(1), cols(other.cols), data(other.data + rowid * other.cols), storage(other.storage) {
    if (rowid >= other.rows) {
        throw std::invalid_argument("row index out of bounds for matrix");
    }
}

Matrix::Matrix(const Matrix & other, int start_rowid, int end_rowid)
    : rows(end_rowid - start_rowid), cols(other.cols), data(other.data + start_rowid * other.cols), storage(other.storage) {
    if (end_rowid < start_rowid) {
        throw std::invalid_argument("end_rowid < start_rowid for matrix slice");
    }
    if (end_rowid > other.rows) {
        throw std::invalid_argument("row index out of bounds for matrix");
    }
}

void copy_rowids(const float * input, const int * rowids, int rows, int cols, float * output) {
    // copy rows over
    auto count = thrust::make_counting_iterator<int>(0);
    thrust::for_each(count, count + (rows * cols),
        [=] __device__(int i) {
            int col = i % cols;
            int row = rowids[i / cols];
            output[i] = input[col + row * cols];
        });
}

Matrix::Matrix(const Matrix & other, const Vector<int> & rowids)
    : rows(rowids.size), cols(other.cols) {
    storage.reset(new DeviceBuffer<float>(rows * cols));
    data = storage->get();
    copy_rowids(other.data, rowids.data, rows, cols, data);
}

Matrix::Matrix(int rows, int cols, float * host_data, bool allocate)
    : rows(rows), cols(cols) {
    if (allocate) {
        storage.reset(new DeviceBuffer<float>(rows * cols));
        data = storage->get();
        if (host_data) {
            CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * sizeof(float), hipMemcpyHostToDevice));
        }
    } else {
        data = host_data;
    }
}

__global__ void calculate_norms_kernel(const float * input, int rows, int cols, float * output) {
    static __shared__ float shared[32];
    for (int i = blockIdx.x; i < rows; i += gridDim.x) {
        float value = input[i * cols + threadIdx.x];
	float squared_norm = dot(value, value, shared);
	if (threadIdx.x == 0) {
            output[i] = sqrt(squared_norm);
            if (output[i] == 0) {
                output[i] = 1e-10;
            }
	}
    }
}

Matrix calculate_norms(const Matrix & input) {
    int devId;
    CHECK_CUDA(hipGetDevice(&devId));

    int multiprocessor_count;
    CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                      hipDeviceAttributeMultiprocessorCount,
                                      devId));

    int block_count = 256 * multiprocessor_count;
    int thread_count = input.cols;

    Matrix output(1, input.rows, NULL);
    calculate_norms_kernel<<<block_count, thread_count>>>(input.data, input.rows, input.cols, output.data);

    CHECK_CUDA(hipDeviceSynchronize());
    return output;
}

void Matrix::to_host(float * out) const {
    CHECK_CUDA(hipMemcpy(out, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
}

CSRMatrix::CSRMatrix(int rows, int cols, int nonzeros,
                     const int * indptr_, const int * indices_, const float * data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

    CHECK_CUDA(hipMalloc(&indptr, (rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1)*sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&indices, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
    CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

CSRMatrix::~CSRMatrix() {
    CHECK_CUDA(hipFree(indices));
    CHECK_CUDA(hipFree(indptr));
    CHECK_CUDA(hipFree(data));
}

COOMatrix::COOMatrix(int rows, int cols, int nonzeros,
                     const int * row_, const int * col_, const float * data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

    CHECK_CUDA(hipMalloc(&row, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(row, row_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&col, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(col, col_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
    CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

COOMatrix::~COOMatrix() {
    CHECK_CUDA(hipFree(row));
    CHECK_CUDA(hipFree(col));
    CHECK_CUDA(hipFree(data));
}
}}  // namespace implicit::gpu
