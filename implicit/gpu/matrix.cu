
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "implicit/gpu/matrix.h"
#include "implicit/gpu/utils.cuh"

namespace implicit {
template <typename T>
CudaVector<T>::CudaVector(int size, const T * host_data)
    : size(size) {
    CHECK_CUDA(hipMalloc(&data, size * sizeof(T)));
    if (host_data) {
        CHECK_CUDA(hipMemcpy(data, host_data, size * sizeof(T), hipMemcpyHostToDevice));
    }
}


template <typename T>
CudaVector<T>::~CudaVector() {
    CHECK_CUDA(hipFree(data));
}

template struct CudaVector<int>;
template struct CudaVector<float>;

CudaDenseMatrix::CudaDenseMatrix(int rows, int cols, float * host_data, bool cpu)
    : rows(rows), cols(cols) {
    if (cpu) {
        CHECK_CUDA(hipMalloc(&data, rows * cols * sizeof(float)));
        if (host_data) {
            CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * sizeof(float), hipMemcpyHostToDevice));
        }
        owns_data = true;
    } else {
        data = host_data;
        owns_data = false;
    }
}
void CudaDenseMatrix::to_host(float * out) const {
    CHECK_CUDA(hipMemcpy(out, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
}

CudaDenseMatrix::~CudaDenseMatrix() {
    if (owns_data) {
        CHECK_CUDA(hipFree(data));
    }
}

CudaCSRMatrix::CudaCSRMatrix(int rows, int cols, int nonzeros,
                             const int * indptr_, const int * indices_, const float * data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

    CHECK_CUDA(hipMalloc(&indptr, (rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1)*sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&indices, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
    CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

CudaCSRMatrix::~CudaCSRMatrix() {
    CHECK_CUDA(hipFree(indices));
    CHECK_CUDA(hipFree(indptr));
    CHECK_CUDA(hipFree(data));
}

CudaCOOMatrix::CudaCOOMatrix(int rows, int cols, int nonzeros,
                             const int * row_, const int * col_, const float * data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

    CHECK_CUDA(hipMalloc(&row, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(row, row_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&col, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(col, col_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
    CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

CudaCOOMatrix::~CudaCOOMatrix() {
    CHECK_CUDA(hipFree(row));
    CHECK_CUDA(hipFree(col));
    CHECK_CUDA(hipFree(data));
}
}  // namespace implicit
