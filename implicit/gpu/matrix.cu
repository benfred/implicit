#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include "implicit/gpu/dot.cuh"
#include "implicit/gpu/matrix.h"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {
template <typename T>
Vector<T>::Vector(int size, const T *host_data)
    : size(size),
      storage(new rmm::device_uvector<T>(size, rmm::cuda_stream_view())),
      data(storage->data()) {
  if (host_data) {
    CHECK_CUDA(
        hipMemcpy(data, host_data, size * sizeof(T), hipMemcpyHostToDevice));
  }
}

template <typename T> void Vector<T>::to_host(T *out) const {
  CHECK_CUDA(hipMemcpy(out, data, size * sizeof(T), hipMemcpyDeviceToHost));
}

template struct Vector<char>;
template struct Vector<int>;
template struct Vector<float>;

Matrix::Matrix(const Matrix &other, int rowid)
    : rows(1), cols(other.cols), data(other.data + rowid * other.cols),
      storage(other.storage) {
  if (rowid >= other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

Matrix::Matrix(const Matrix &other, int start_rowid, int end_rowid)
    : rows(end_rowid - start_rowid), cols(other.cols),
      data(other.data + start_rowid * other.cols), storage(other.storage) {
  if (end_rowid < start_rowid) {
    throw std::invalid_argument("end_rowid < start_rowid for matrix slice");
  }
  if (end_rowid > other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

void copy_rowids(const float *input, const int *rowids, int rows, int cols,
                 float *output) {
  // copy rows over
  auto count = thrust::make_counting_iterator<int>(0);
  thrust::for_each(count, count + (rows * cols), [=] __device__(int i) {
    int col = i % cols;
    int row = rowids[i / cols];
    output[i] = input[col + row * cols];
  });
}

Matrix::Matrix(const Matrix &other, const Vector<int> &rowids)
    : rows(rowids.size), cols(other.cols) {
  storage.reset(
      new rmm::device_uvector<float>(rows * cols, rmm::cuda_stream_view()));
  data = storage->data();
  copy_rowids(other.data, rowids.data, rows, cols, data);
}

Matrix::Matrix(int rows, int cols, float *host_data, bool allocate)
    : rows(rows), cols(cols) {
  if (allocate) {
    storage.reset(
        new rmm::device_uvector<float>(rows * cols, rmm::cuda_stream_view()));
    data = storage->data();
    if (host_data) {
      CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * sizeof(float),
                            hipMemcpyHostToDevice));
    }
  } else {
    data = host_data;
  }
}

void Matrix::resize(int rows, int cols) {
  if (cols != this->cols) {
    throw std::logic_error(
        "changing number of columns in Matrix::resize is not implemented yet");
  }
  if (rows < this->rows) {
    throw std::logic_error(
        "reducing number of rows in Matrix::resize is not implemented yet");
  }
  auto new_storage =
      new rmm::device_uvector<float>(rows * cols, rmm::cuda_stream_view());
  CHECK_CUDA(hipMemcpy(new_storage->data(), data,
                        this->rows * this->cols * sizeof(float),
                        hipMemcpyDeviceToDevice));
  int extra_rows = rows - this->rows;
  CHECK_CUDA(hipMemset(new_storage->data() + this->rows * this->cols, 0,
                        extra_rows * cols * sizeof(float)));
  storage.reset(new_storage);
  data = storage->data();
  this->rows = rows;
  this->cols = cols;
}

void Matrix::assign_rows(const Vector<int> &rowids, const Matrix &other) {
  if (other.cols != cols) {
    throw std::invalid_argument(
        "column dimensionality mismatch in Matrix::assign_rows");
  }

  auto count = thrust::make_counting_iterator<int>(0);
  int other_cols = other.cols, other_rows = other.rows;

  int *rowids_data = rowids.data;
  float *other_data = other.data;
  float *self_data = data;

  thrust::for_each(count, count + (other_rows * other_cols),
                   [=] __device__(int i) {
                     int col = i % other_cols;
                     int row = rowids_data[i / other_cols];
                     int idx = col + row * other_cols;
                     self_data[idx] = other_data[i];
                   });
}

__global__ void calculate_norms_kernel(const float *input, int rows, int cols,
                                       float *output) {
  static __shared__ float shared[32];
  for (int i = blockIdx.x; i < rows; i += gridDim.x) {
    float value = input[i * cols + threadIdx.x];
    float squared_norm = dot(value, value, shared);
    if (threadIdx.x == 0) {
      output[i] = sqrt(squared_norm);
      if (output[i] == 0) {
        output[i] = 1e-10;
      }
    }
  }
}

Matrix calculate_norms(const Matrix &input) {
  int devId;
  CHECK_CUDA(hipGetDevice(&devId));

  int multiprocessor_count;
  CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                    hipDeviceAttributeMultiprocessorCount, devId));

  int block_count = 256 * multiprocessor_count;
  int thread_count = input.cols;

  Matrix output(1, input.rows, NULL);
  calculate_norms_kernel<<<block_count, thread_count>>>(
      input.data, input.rows, input.cols, output.data);

  CHECK_CUDA(hipDeviceSynchronize());
  return output;
}

void Matrix::to_host(float *out) const {
  CHECK_CUDA(hipMemcpy(out, data, rows * cols * sizeof(float),
                        hipMemcpyDeviceToHost));
}

CSRMatrix::CSRMatrix(int rows, int cols, int nonzeros, const int *indptr_,
                     const int *indices_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMalloc(&indptr, (rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&indices, nonzeros * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int),
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(
      hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

CSRMatrix::~CSRMatrix() {
  CHECK_CUDA(hipFree(indices));
  CHECK_CUDA(hipFree(indptr));
  CHECK_CUDA(hipFree(data));
}

COOMatrix::COOMatrix(int rows, int cols, int nonzeros, const int *row_,
                     const int *col_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMalloc(&row, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(row, row_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&col, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(col, col_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(
      hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

COOMatrix::~COOMatrix() {
  CHECK_CUDA(hipFree(row));
  CHECK_CUDA(hipFree(col));
  CHECK_CUDA(hipFree(data));
}
} // namespace gpu
} // namespace implicit
