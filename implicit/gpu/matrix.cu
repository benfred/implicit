#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include "implicit/gpu/convert.cuh"
#include "implicit/gpu/dot.cuh"
#include "implicit/gpu/matrix.h"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {
template <typename T>
Vector<T>::Vector(size_t size, const T *host_data)
    : size(size),
      storage(new rmm::device_uvector<T>(size, rmm::cuda_stream_view())),
      data(storage->data()) {
  if (host_data) {
    CHECK_CUDA(
        hipMemcpy(data, host_data, size * sizeof(T), hipMemcpyHostToDevice));
  } else {
    CHECK_CUDA(hipMemset(data, 0, size * sizeof(T)));
  }
}

template <typename T> void Vector<T>::to_host(T *out) const {
  CHECK_CUDA(hipMemcpy(out, data, size * sizeof(T), hipMemcpyDeviceToHost));
}

template struct Vector<char>;
template struct Vector<int>;
template struct Vector<float>;

Matrix::Matrix(const Matrix &other, size_t rowid)
    : rows(1), cols(other.cols), data(other.at(rowid * other.cols)),
      storage(other.storage), itemsize(other.itemsize) {
  if (rowid >= other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

Matrix::Matrix(const Matrix &other, size_t start_rowid, size_t end_rowid)
    : rows(end_rowid - start_rowid), cols(other.cols),
      data(other.at(start_rowid * other.cols)), storage(other.storage),
      itemsize(other.itemsize) {
  if (end_rowid < start_rowid) {
    throw std::invalid_argument("end_rowid < start_rowid for matrix slice");
  }
  if (end_rowid > other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

template <typename T>
void copy_rowids(const T *input, const int *rowids, size_t rows, size_t cols,
                 T *output) {
  // copy rows over
  auto count = thrust::make_counting_iterator<size_t>(0);
  thrust::for_each(count, count + (rows * cols), [=] __device__(size_t i) {
    size_t col = i % cols;
    size_t row = rowids[i / cols];
    output[i] = input[col + row * cols];
  });
}

Matrix::Matrix(const Matrix &other, const Vector<int> &rowids)
    : rows(rowids.size), cols(other.cols), itemsize(other.itemsize) {
  storage.reset(
      new rmm::device_buffer(itemsize * rows * cols, rmm::cuda_stream_view()));
  data = storage->data();
  if (itemsize == 4) {
    copy_rowids<float>(other, rowids.data, rows, cols, *this);
  } else if (itemsize == 2) {
    copy_rowids<half>(other, rowids.data, rows, cols, *this);
  } else {
    throw std::runtime_error("unknown itemsize initializing Matrix");
  }
}

Matrix::Matrix(size_t rows, size_t cols, void *host_data, bool allocate,
               size_t itemsize)
    : rows(rows), cols(cols), itemsize(itemsize) {
  if (allocate) {
    storage.reset(new rmm::device_buffer(itemsize * rows * cols,
                                         rmm::cuda_stream_view()));
    data = storage->data();
    if (host_data) {
      CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * itemsize,
                            hipMemcpyHostToDevice));
    } else {
      CHECK_CUDA(hipMemset(data, 0, rows * cols * itemsize));
    }
  } else {
    data = host_data;
  }
}

void Matrix::resize(size_t rows, size_t cols) {
  if (cols != this->cols) {
    throw std::logic_error(
        "changing number of columns in Matrix::resize is not implemented yet");
  }
  if (rows < this->rows) {
    throw std::logic_error(
        "reducing number of rows in Matrix::resize is not implemented yet");
  }
  auto new_storage =
      new rmm::device_buffer(itemsize * rows * cols, rmm::cuda_stream_view());
  CHECK_CUDA(hipMemcpy(new_storage->data(), data,
                        this->rows * this->cols * itemsize,
                        hipMemcpyDeviceToDevice));
  size_t extra_rows = rows - this->rows;
  storage.reset(new_storage);
  data = storage->data();
  CHECK_CUDA(
      hipMemset(at(this->rows * this->cols), 0, extra_rows * cols * itemsize));

  this->rows = rows;
  this->cols = cols;
}

void Matrix::assign_rows(const Vector<int> &rowids, const Matrix &other) {
  if (other.cols != cols) {
    throw std::invalid_argument(
        "column dimensionality mismatch in Matrix::assign_rows");
  }

  auto count = thrust::make_counting_iterator<size_t>(0);
  size_t other_cols = other.cols, other_rows = other.rows;

  int *rowids_data = rowids.data;

  const float *other_data = other;
  float *self_data = *this;

  thrust::for_each(count, count + (other_rows * other_cols),
                   [=] __device__(size_t i) {
                     size_t col = i % other_cols;
                     size_t row = rowids_data[i / other_cols];
                     size_t idx = col + row * other_cols;
                     self_data[idx] = other_data[i];
                   });
}

template <typename I, typename O>
void convert_array(const I *input, size_t elements, O *output) {
  auto count = thrust::make_counting_iterator<size_t>(0);
  thrust::for_each(count, count + elements, [=] __device__(size_t i) {
    output[i] = convert<I, O>(input[i]);
  });
}

Matrix Matrix::astype(size_t itemsize) const {
  if (itemsize == this->itemsize) {
    return *this;
  }

  Matrix ret(rows, cols, NULL, true, itemsize);
  if (this->itemsize == 2 && itemsize == 4) {
    const half *input_data = *this;
    float *output_data = ret;
    convert_array<half, float>(input_data, rows * cols, output_data);
  } else if (this->itemsize == 4 && itemsize == 2) {
    const float *input_data = *this;
    half *output_data = ret;
    convert_array<float, half>(input_data, rows * cols, output_data);
  } else {
    throw std::runtime_error("Invalid dtype for Matrix::astype");
  }
  return ret;
}

template <typename T>
__global__ void calculate_norms_kernel(const T *input, size_t rows, size_t cols,
                                       float *output) {
  static __shared__ float shared[32];
  for (int i = blockIdx.x; i < rows; i += gridDim.x) {
    float value = convert<T, float>(input[i * cols + threadIdx.x]);
    float squared_norm = dot(value, value, shared);
    if (threadIdx.x == 0) {
      float norm = sqrt(squared_norm);
      if (norm == 0) {
        norm = 1e-10;
      }
      output[i] = norm;
    }
  }
}

Matrix Matrix::calculate_norms() const {
  int devId;
  CHECK_CUDA(hipGetDevice(&devId));

  int multiprocessor_count;
  CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                    hipDeviceAttributeMultiprocessorCount, devId));

  int block_count = 256 * multiprocessor_count;
  int thread_count = cols;

  Matrix output(1, rows, NULL, true);

  if (itemsize == 4) {
    calculate_norms_kernel<float>
        <<<block_count, thread_count>>>(*this, rows, cols, output);
  } else if (itemsize == 2) {
    calculate_norms_kernel<half>
        <<<block_count, thread_count>>>(*this, rows, cols, output);
  } else {
    throw std::runtime_error("unknown itemsize in calculate_norms");
  }

  CHECK_CUDA(hipDeviceSynchronize());
  return output;
}

void Matrix::to_host(void *out) const {
  CHECK_CUDA(
      hipMemcpy(out, data, rows * cols * itemsize, hipMemcpyDeviceToHost));
}

CSRMatrix::CSRMatrix(int rows, int cols, int nonzeros, const int *indptr_,
                     const int *indices_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMallocManaged(&indptr, (rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemAdvise(indptr, (rows + 1) * sizeof(int),
                           hipMemAdviseSetReadMostly, 0));

  CHECK_CUDA(hipMallocManaged(&indices, nonzeros * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemAdvise(indices, nonzeros * sizeof(int),
                           hipMemAdviseSetReadMostly, 0));

  CHECK_CUDA(hipMallocManaged(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemAdvise(data, nonzeros * sizeof(float),
                           hipMemAdviseSetReadMostly, 0));
}

CSRMatrix::~CSRMatrix() {
  CHECK_CUDA(hipFree(indices));
  CHECK_CUDA(hipFree(indptr));
  CHECK_CUDA(hipFree(data));
}

COOMatrix::COOMatrix(int rows, int cols, int nonzeros, const int *row_,
                     const int *col_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMallocManaged(&row, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(row, row_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMallocManaged(&col, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(col, col_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMallocManaged(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(
      hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

COOMatrix::~COOMatrix() {
  CHECK_CUDA(hipFree(row));
  CHECK_CUDA(hipFree(col));
  CHECK_CUDA(hipFree(data));
}
} // namespace gpu
} // namespace implicit
