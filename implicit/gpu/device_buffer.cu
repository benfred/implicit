#include <hip/hip_runtime.h>
#include <iostream>

#include "implicit/gpu/utils.cuh"
#include "implicit/gpu/device_buffer.h"

namespace implicit { namespace gpu {

template <typename T>
DeviceBuffer<T>::DeviceBuffer(size_t size) : size_(size) {
    // TODO: support custom allocators (rmm etc) ?
    CHECK_CUDA(hipMalloc(&data, size * sizeof(T)));
}

template <typename T>
DeviceBuffer<T>::~DeviceBuffer() {
    auto err = hipFree(data);
    if (err != hipSuccess) {
        std::cerr << "Failed to call hipFree in ~DeviceBuffer:" << hipGetErrorString(err) << std::endl;
    }
}

template <typename T>
DeviceBuffer<T>::DeviceBuffer(DeviceBuffer<T> && other) {
    if (this != &other) {
        data = other.data;
        size_ = other.size_;
        other.data = NULL;
        other.size_ = 0;
    }
}

template struct DeviceBuffer<int>;
template struct DeviceBuffer<float>;
template struct DeviceBuffer<char>;
}}  // namespace implicit::gpu
