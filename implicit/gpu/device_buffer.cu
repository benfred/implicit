#include <hip/hip_runtime.h>
#include <iostream>

#include "implicit/gpu/device_buffer.h"
#include "implicit/gpu/utils.cuh"

namespace implicit {
namespace gpu {

template <typename T> DeviceBuffer<T>::DeviceBuffer(size_t size) : size_(size) {
  // TODO: support custom allocators (rmm etc) ?
  CHECK_CUDA(hipMalloc(&data, size * sizeof(T)));
}

template <typename T> DeviceBuffer<T>::~DeviceBuffer() {
  auto err = hipFree(data);
  if (err != hipSuccess) {
    std::cerr << "Failed to call hipFree in ~DeviceBuffer:"
              << hipGetErrorString(err) << std::endl;
  }
}

template <typename T> DeviceBuffer<T>::DeviceBuffer(DeviceBuffer<T> &&other) {
  if (this != &other) {
    data = other.data;
    size_ = other.size_;
    other.data = NULL;
    other.size_ = 0;
  }
}

template struct DeviceBuffer<int>;
template struct DeviceBuffer<float>;
template struct DeviceBuffer<char>;
} // namespace gpu
} // namespace implicit
