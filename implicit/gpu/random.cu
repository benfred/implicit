#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>

#include "implicit/gpu/utils.cuh"
#include "implicit/gpu/random.h"

namespace implicit { namespace gpu {

RandomState::RandomState(long seed) {
    CHECK_CURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(rng, seed));
}

Matrix RandomState::uniform(int rows, int cols, float low, float high) {
    Matrix ret(rows, cols, NULL);
    CHECK_CURAND(hiprandGenerateUniform(rng, ret.data, rows*cols));

    if ((low != 0.0) || (high != 1.0)) {
        auto start = thrust::device_pointer_cast(ret.data);
        thrust::transform(start, start + rows*cols, start, 
            thrust::placeholders::_1 = thrust::placeholders::_1 * (high - low) + low);
    }

    return ret;
}

Matrix RandomState::randn(int rows, int cols, float mean, float stddev) {
    Matrix ret(rows, cols, NULL);
    CHECK_CURAND(hiprandGenerateNormal(rng, ret.data, rows*cols, mean, stddev));
    return ret;
}

RandomState::~RandomState() {
    hiprandDestroyGenerator(rng);
}
}}  // namespace implicit::gpu
