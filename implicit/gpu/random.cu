#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "implicit/gpu/random.h"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {

RandomState::RandomState(long seed) {
  CHECK_CURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
  CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(rng, seed));
}

Matrix RandomState::uniform(size_t rows, size_t cols, float low, float high) {
  Matrix ret(rows, cols, NULL);
  CHECK_CURAND(hiprandGenerateUniform(rng, ret, rows * cols));

  if ((low != 0.0) || (high != 1.0)) {
    float *data = ret;
    auto start = thrust::device_pointer_cast(data);
    thrust::transform(start, start + rows * cols, start,
                      thrust::placeholders::_1 =
                          thrust::placeholders::_1 * (high - low) + low);
  }

  return ret;
}

Matrix RandomState::randn(size_t rows, size_t cols, float mean, float stddev) {
  Matrix ret(rows, cols, NULL);
  CHECK_CURAND(hiprandGenerateNormal(rng, ret, rows * cols, mean, stddev));
  return ret;
}

RandomState::~RandomState() { hiprandDestroyGenerator(rng); }
} // namespace gpu
} // namespace implicit
