#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "implicit/gpu/als.h"
#include "implicit/gpu/dot.cuh"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {

using std::invalid_argument;

__global__ void least_squares_cg_kernel(int factors, size_t user_count,
                                        size_t item_count, float *X,
                                        const float *Y, const float *YtY,
                                        const int *indptr, const int *indices,
                                        const float *data, int cg_steps) {
  extern __shared__ float shared_memory[];
  float *P = &shared_memory[0];
  float *shared = &shared_memory[factors];

  float Ap = 0;
  float p = 0;
  float r = 0;

  // Stride over users in the grid:
  // https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
  for (int u = blockIdx.x; u < user_count; u += gridDim.x) {
    float *x = &X[u * factors];
    float x_value = x[threadIdx.x];

    // handle 0-sized rows
    if (indptr[u] == indptr[u + 1]) {
      x[threadIdx.x] = 0;
      continue;
    }

    // calculate residual r = YtCuPu - YtCuY Xu
    r = 0;
    for (int i = 0; i < factors; ++i) {
      r -= x[i] * YtY[i * factors + threadIdx.x];
    }
    for (int index = indptr[u]; index < indptr[u + 1]; ++index) {
      float Yi = Y[indices[index] * factors + threadIdx.x];
      float confidence = data[index];

      if (confidence > 0) {
        r += (confidence - (confidence - 1) * dot(Yi, x_value, shared)) * Yi;
      } else {
        confidence *= -1;
        r += (-(confidence - 1) * dot(Yi, x_value, shared)) * Yi;
      }
    }
    P[threadIdx.x] = p = r;
    __syncthreads();

    float rsold = dot(r, r, shared);
    if (rsold < 1e-20)
      continue;

    for (int it = 0; it < cg_steps; ++it) {
      // calculate Ap = YtCuYp - without actually calculating YtCuY
      Ap = 0;
      for (int i = 0; i < factors; ++i) {
        Ap += P[i] * YtY[i * factors + threadIdx.x];
      }
      for (int index = indptr[u]; index < indptr[u + 1]; ++index) {
        float Yi = Y[indices[index] * factors + threadIdx.x];
        float confidence = data[index];
        if (confidence < 0)
          confidence *= -1;

        Ap += (confidence - 1) * dot(Yi, p, shared) * Yi;
      }

      // standard CG update
      float alpha = rsold / dot(p, Ap, shared);
      x_value += alpha * p;
      r -= alpha * Ap;
      __syncthreads();
      float rsnew = dot(r, r, shared);
      if (rsnew < 1e-20)
        break;

      P[threadIdx.x] = p = r + (rsnew / rsold) * p;
      rsold = rsnew;
      __syncthreads();
    }

    // this shouldn't happen - but if we hit a NaN in the above code then
    // complain and don't let it perpetuate
    if (isnan(rsold)) {
      if (threadIdx.x == 0) {
        printf("Warning NaN Detected in row %i of %lu\n", u, user_count);
      }
      x[threadIdx.x] = 0;
    } else {
      x[threadIdx.x] = x_value;
    }
  }
}

__global__ void l2_regularize_kernel(size_t factors, float regularization,
                                     float *YtY) {
  YtY[threadIdx.x * factors + threadIdx.x] += regularization;
}

LeastSquaresSolver::LeastSquaresSolver() {
  CHECK_CUBLAS(hipblasCreate(&blas_handle));
}

void LeastSquaresSolver::calculate_yty(const Matrix &Y, Matrix *YtY,
                                       float regularization) {
  if (YtY->cols != Y.cols)
    throw invalid_argument("YtY and Y should have the same number of columns");

  // calculate YtY: note this expects col-major (and we have row-major
  // basically) so that we're inverting the HIPBLAS_OP_T/CU_BLAS_OP_N ordering to
  // overcome this (like calculate YYt instead of YtY)
  size_t factors = Y.cols, item_count = Y.rows;
  float alpha = 1.0, beta = 0.;
  CHECK_CUBLAS(hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, factors,
                           factors, item_count, &alpha, Y.data, factors, Y.data,
                           factors, &beta, YtY->data, factors));
  CHECK_CUDA(hipDeviceSynchronize());

  // regularize the matrix
  l2_regularize_kernel<<<1, factors>>>(factors, regularization, YtY->data);
  CHECK_CUDA(hipDeviceSynchronize());
}

void LeastSquaresSolver::least_squares(const CSRMatrix &Cui, Matrix *X,
                                       const Matrix &YtY, const Matrix &Y,
                                       int cg_steps) const {
  int item_count = Y.rows, user_count = X->rows, factors = X->cols;
  if (X->cols != Y.cols)
    throw invalid_argument("X and Y should have the same number of columns");
  if (X->cols != YtY.cols)
    throw invalid_argument("Columns of X don't match number of factors");
  if (Cui.rows > X->rows)
    throw invalid_argument("Dimensionality mismatch between rows of Cui and X");
  if (Cui.cols > Y.rows)
    throw invalid_argument("Dimensionality mismatch between cols of Cui and Y");

  // TODO: multi-gpu support
  int devId;
  CHECK_CUDA(hipGetDevice(&devId));

  int multiprocessor_count;
  CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                    hipDeviceAttributeMultiprocessorCount, devId));

  int block_count = 256 * multiprocessor_count;
  int thread_count = factors;
  int shared_memory_size = sizeof(float) * (2 * factors);

  least_squares_cg_kernel<<<block_count, thread_count, shared_memory_size>>>(
      factors, user_count, item_count, X->data, Y.data, YtY.data, Cui.indptr,
      Cui.indices, Cui.data, cg_steps);

  CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void calculate_loss_kernel(int factors, size_t user_count,
                                      size_t item_count, const float *X,
                                      const float *Y, const float *YtY,
                                      const int *indptr, const int *indices,
                                      const float *data, float regularization,
                                      float *output) {
  // https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
  extern __shared__ float shared_memory[];
  float *shared = &shared_memory[0];

  float loss = 0, user_norm = 0, item_norm = 0, total_confidence = 0, r = 0;

  for (int u = blockIdx.x; u < user_count; u += gridDim.x) {
    const float *x = &X[u * factors];
    float x_value = x[threadIdx.x];

    // calculates r = (YtCuY.dot(Xu) - 2 * YtCuPu).dot(Xu), without calculating
    // YtCuY
    r = 0;
    for (int i = 0; i < factors; ++i) {
      r += x[i] * YtY[i * factors + threadIdx.x];
    }

    for (int index = indptr[u]; index < indptr[u + 1]; ++index) {
      float Yi = Y[indices[index] * factors + threadIdx.x];
      float confidence = data[index];
      if (confidence > 0) {
        r +=
            ((confidence - 1) * dot(Yi, x_value, shared) - 2 * confidence) * Yi;
      } else {
        confidence *= -1;
        r += ((confidence - 1) * dot(Yi, x_value, shared)) * Yi;
      }
      loss += confidence;
      total_confidence += confidence;
    }
    loss += dot(x_value, r, shared);

    user_norm += dot(x_value, x_value, shared);
  }

  for (int i = blockIdx.x; i < item_count; i += gridDim.x) {
    float y = Y[i * factors + threadIdx.x];
    item_norm += dot(y, y, shared);
  }

  loss += regularization * (item_norm + user_norm);
  if (threadIdx.x == 0) {
    atomicAdd(output, loss);
    atomicAdd(output + 1, total_confidence);
  }
}

float LeastSquaresSolver::calculate_loss(const CSRMatrix &Cui, const Matrix &X,
                                         const Matrix &Y,
                                         float regularization) {
  size_t item_count = Y.rows, factors = Y.cols, user_count = X.rows;

  Matrix YtY(factors, factors, NULL);
  calculate_yty(Y, &YtY, regularization);

  float alpha = 1.0, beta = 0.;
  CHECK_CUBLAS(hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, factors,
                           factors, item_count, &alpha, Y.data, factors, Y.data,
                           factors, &beta, YtY.data, factors));
  CHECK_CUDA(hipDeviceSynchronize());
  float temp[2] = {0, 0};
  Matrix output(2, 1, temp);
  calculate_loss_kernel<<<1024, factors, sizeof(float) * factors>>>(
      factors, user_count, item_count, X.data, Y.data, YtY.data, Cui.indptr,
      Cui.indices, Cui.data, regularization, output.data);
  CHECK_CUDA(hipDeviceSynchronize());
  output.to_host(temp);

  return temp[0] / (temp[1] + Cui.rows * Cui.cols - Cui.nonzeros);
}

LeastSquaresSolver::~LeastSquaresSolver() {
  CHECK_CUBLAS(hipblasDestroy(blas_handle));
}
} // namespace gpu
} // namespace implicit
