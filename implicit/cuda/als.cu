#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "implicit/cuda/als.h"
#include "implicit/cuda/utils.cuh"

namespace implicit {

using std::invalid_argument;

__global__ void least_squares_cg_kernel(int factors, int user_count, int item_count,
                                        float * X, const float * Y, const float * YtY,
                                        const int * indptr, const int * indices,
                                        const float * data, int cg_steps) {
    // Ap/r/p are vectors for CG update - use dynamic shared memory to store
    // https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
    extern __shared__ float shared_memory[];
    float * Ap = &shared_memory[0];
    float * r = &shared_memory[factors];
    float * p = &shared_memory[2*factors];

    // Stride over users in the grid:
    // https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    for (int u = blockIdx.x; u < user_count; u += gridDim.x) {
        float * x = &X[u * factors];

        // handle 0-sized rows
        if (indptr[u] == indptr[u+1]) {
            x[threadIdx.x] = 0;
            continue;
        }

        // calculate residual r = YtCuPu - YtCuY Xu
        float temp = 0;
        for (int i = 0; i < factors; ++i) {
            temp -= x[i] * YtY[i * factors + threadIdx.x];
        }
        for (int index = indptr[u]; index < indptr[u + 1]; ++index) {
            const float * Yi = &Y[indices[index] * factors];
            float confidence = data[index];

            if (confidence > 0) {
                temp += (confidence - (confidence - 1) * dot(Yi, x)) * Yi[threadIdx.x];
            } else {
                confidence *= -1;
                temp += (- (confidence - 1) * dot(Yi, x)) * Yi[threadIdx.x];
            }
        }
        p[threadIdx.x] = r[threadIdx.x] = temp;

        float rsold = dot(r, r);
        if (rsold < 1e-20) continue;

        for (int it = 0; it < cg_steps; ++it) {
            // calculate Ap = YtCuYp - without actually calculating YtCuY
            Ap[threadIdx.x] = 0;
            for (int i = 0; i < factors; ++i) {
                Ap[threadIdx.x] += p[i] * YtY[i * factors + threadIdx.x];
            }
            for (int index = indptr[u]; index < indptr[u + 1]; ++index) {
                const float * Yi = &Y[indices[index] * factors];
                float confidence = data[index];
                if (confidence < 0) confidence *= -1;

                Ap[threadIdx.x] += (confidence - 1) * dot(Yi, p) * Yi[threadIdx.x];
            }

            // standard CG update
            float alpha = rsold / dot(p, Ap);
            x[threadIdx.x] += alpha * p[threadIdx.x];
            r[threadIdx.x] -= alpha * Ap[threadIdx.x];
            float rsnew = dot(r, r);
            if (rsnew < 1e-20) break;
            p[threadIdx.x] = r[threadIdx.x] + (rsnew/rsold) * p[threadIdx.x];
            rsold = rsnew;
            __syncthreads();
        }

        // this shouldn't happen - but if we hit a NaN in the above code then complain
        // and don't let it perpetuate
        if (isnan(rsold)) {
            if (threadIdx.x == 0) {
                printf("Warning NaN Detected in row %d of %d\n", u, user_count);
            }
            x[threadIdx.x] = 0;
        }
    }
}

__global__ void l2_regularize_kernel(int factors, float regularization, float * YtY) {
    YtY[threadIdx.x * factors + threadIdx.x] += regularization;
}
CudaLeastSquaresSolver::CudaLeastSquaresSolver(int factors)
    : YtY(factors, factors, NULL) {
    CHECK_CUBLAS(hipblasCreate(&blas_handle));
}

void CudaLeastSquaresSolver::least_squares(const CudaCSRMatrix & Cui,
                                           CudaDenseMatrix * X,
                                           const CudaDenseMatrix & Y,
                                           float regularization,
                                           int cg_steps) const {
    int item_count = Y.rows, user_count = X->rows, factors = X->cols;
    if (X->cols != Y.cols) throw invalid_argument("X and Y should have the same number of columns");
    if (X->cols != YtY.cols) throw invalid_argument("Columns of X don't match number of factors");
    if (Cui.rows != X->rows) throw invalid_argument("Dimensionality mismatch between Cui and X");
    if (Cui.cols != Y.rows) throw invalid_argument("Dimensionality mismatch between Cui and Y");

    // calculate YtY: note this expects col-major (and we have row-major basically)
    // so that we're inverting the HIPBLAS_OP_T/CU_BLAS_OP_N ordering to overcome
    // this (like calculate YYt instead of YtY)
    float alpha = 1.0, beta = 0.;
    CHECK_CUBLAS(hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                             factors, factors, item_count,
                             &alpha,
                             Y.data, factors,
                             Y.data, factors,
                             &beta,
                             YtY.data, factors));
    CHECK_CUDA(hipDeviceSynchronize());

    // regularize the matrix
    l2_regularize_kernel<<<1, factors>>>(factors, regularization, YtY.data);
    CHECK_CUDA(hipDeviceSynchronize());

    // TODO: multi-gpu support
    int devId;
    CHECK_CUDA(hipGetDevice(&devId));

    int multiprocessor_count;
    CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                      hipDeviceAttributeMultiprocessorCount,
                                      devId));

    int block_count = 128 * multiprocessor_count;
    int thread_count = factors;
    int shared_memory_size = sizeof(float) * (3 * factors);

    least_squares_cg_kernel<<<block_count, thread_count, shared_memory_size>>>(
        factors, user_count, item_count,
        X->data, Y.data, YtY.data, Cui.indptr, Cui.indices, Cui.data, cg_steps);

    CHECK_CUDA(hipDeviceSynchronize());
}

__global__
void calculate_loss_kernel(int factors, int user_count, int item_count,
                           const float * X, const float * Y, const float * YtY,
                           const int * indptr, const int * indices,
                           const float * data, float regularization, float * output) {
    // https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
    extern __shared__ float shared_memory[];
    float * r = &shared_memory[0];

    float loss = 0, user_norm = 0, item_norm = 0, total_confidence = 0;

    for (int u = blockIdx.x; u < user_count; u += gridDim.x) {
        const float * x = &X[u * factors];

        // calculates r = (YtCuY.dot(Xu) - 2 * YtCuPu).dot(Xu), without calculating YtCuY
        float temp = 0;
        for (int i = 0; i < factors; ++i) {
            temp += x[i] * YtY[i * factors + threadIdx.x];
        }

        for (int index = indptr[u]; index < indptr[u + 1]; ++index) {
            const float * Yi = &Y[indices[index] * factors];
            float confidence = data[index];
            if (confidence > 0) {
                temp += ((confidence - 1 ) * dot(Yi, x) - 2 * confidence) * Yi[threadIdx.x];
            } else {
                confidence *= -1;
                temp += ((confidence - 1 ) * dot(Yi, x)) * Yi[threadIdx.x];
            }
            loss += confidence;
            total_confidence += confidence;
        }
        r[threadIdx.x] = temp;
        loss += dot(x, r);

        user_norm += dot(x, x);
    }
    for (int i = blockIdx.x; i < item_count; i += gridDim.x) {
        const float * y = &Y[i * factors];
        item_norm += dot(y, y);
    }

    loss += regularization * (item_norm + user_norm);
    if (threadIdx.x == 0) {
        atomicAdd(output, loss);
        atomicAdd(output + 1, total_confidence);
    }
}

float CudaLeastSquaresSolver::calculate_loss(const CudaCSRMatrix & Cui,
                                            const CudaDenseMatrix & X,
                                            const CudaDenseMatrix & Y,
                                            float regularization) {
    int item_count = Y.rows, factors = Y.cols, user_count = X.rows;

    float alpha = 1.0, beta = 0.;
    CHECK_CUBLAS(hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                             factors, factors, item_count,
                             &alpha,
                             Y.data, factors,
                             Y.data, factors,
                             &beta,
                             YtY.data, factors));
    CHECK_CUDA(hipDeviceSynchronize());
    float temp[2] = {0, 0};
    CudaDenseMatrix output(2, 1, temp);
    calculate_loss_kernel<<<1024, factors, sizeof(float) * factors>>>(
        factors, user_count, item_count, X.data, Y.data, YtY.data,
        Cui.indptr, Cui.indices, Cui.data, regularization, output.data);
    CHECK_CUDA(hipDeviceSynchronize());
    output.to_host(temp);

    return temp[0] / (temp[1] + Cui.rows * Cui.cols - Cui.nonzeros);
}

CudaLeastSquaresSolver::~CudaLeastSquaresSolver() {
    CHECK_CUBLAS(hipblasDestroy(blas_handle));
}
}  // namespace implicit
